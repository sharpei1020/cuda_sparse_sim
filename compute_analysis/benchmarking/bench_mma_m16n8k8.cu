#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
__forceinline__ __device__ unsigned lane_id()
{
    unsigned ret;
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__forceinline__ __device__ unsigned warp_id()
{
    // this is not equal to threadIdx.x / 32
    unsigned ret;
    asm volatile ("mov.u32 %0, %warpid;" : "=r"(ret));
    return ret;
}

// #define SHARED_MEM_SIZE (32 * 1024 / 4) // 32 KB
// Launch only one thread to calcaulte the latency using a pointer-chasing
// array technique
//#define THREADS_NUM 32
// iterate over the array ITERS times
#ifndef ITERS
#define ITERS  (1024 )
#endif




#ifndef ILPconfig
#define ILPconfig 1
#endif


#if ILPconfig > 6
static_assert(0,"ILP > 6 is not supported\n");
#endif


__global__ void mma_ubench(uint64_t *startClk, uint64_t *stopClk, float *a, float *b, float *res,
          uint32_t strid) { // strid set to 0 used to prevent optimization
  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + tid;
  uint32_t warpid = gid / warpSize;

  a = a + warpid * 16*4; // m*k = 16*16
  b = b + warpid * 8*4; // n*k = 8*16
  res = res + warpid * 16*8;// m*n = 16*16

   /** step 1: create register for each thread **/
  float frag_A[4*ILPconfig]; // two .f16x2 registers, 8 half elements, 
  float frag_B[2*ILPconfig];  // one .f16x2 registers, 4 half  elements
  float frag_D[4*ILPconfig]; //result(fp32) 4 f32 registers

  // fake load, we are focusing on mma latency/throughput. So no need to care about loading
  for(int i = 0;i<4*ILPconfig;i++){
    frag_A[i] = a[i + lane_id()*4]; 
    //frag_A_ILP2[i] = a[i + lane_id()*4] + 1; 
    
  }
  for(int i =0;i<2*ILPconfig;i++){
    frag_B[i] = b[i + lane_id()*1]; 
    //frag_B_ILP2[i] = b[i + lane_id()*1] + 1; 
  }


  for(int i =0;i<4*ILPconfig;i++){
    //frag_B[i] = b[i + lane_id()*4]; 
    frag_D[i] = 0.0f;
    //frag_D_ILP2[i] = 0.0f;
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
  float *C = reinterpret_cast<float *>(&frag_D[0]);
  float *D = C;  // D = A*B + D. 


  // float fpuA = frag_A[0];
  // float fpuB = frag_B[0];
  float fpuC = frag_D[0];

  // int intA = threadIdx.x;
  // int intB = threadIdx.x + 1;
  int intC = threadIdx.x + 2;

  uint64_t start = 0;
  uint64_t stop = 0;
  // synchronize all threads
  asm volatile("bar.sync 0;");
  // start timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");
  //#pragma unroll
  for (int j = 0; j < ITERS; ++j) {
    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]),
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
    );

    #if ILPconfig >= 2
    asm volatile(
      "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[4]), "=f"(D[5]), "=f"(D[6]), "=f"(D[7])
      : "r"(A[2]), "r"(A[3]), 
        "r"(B[1]), 
        "f"(C[4]), "f"(C[5]), "f"(C[6]), "f"(C[7])
    );
    #endif

   #if ILPconfig >= 3
    asm volatile(
      "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[8]), "=f"(D[9]), "=f"(D[10]), "=f"(D[11])
      : "r"(A[4]), "r"(A[5]), 
        "r"(B[2]), 
        "f"(C[8]), "f"(C[9]), "f"(C[10]), "f"(C[11])
    );
    #endif

  #if ILPconfig >= 4
  asm volatile(
    "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
    : "=f"(D[12]), "=f"(D[13]), "=f"(D[14]), "=f"(D[15])
    : "r"(A[6]), "r"(A[7]), 
      "r"(B[3]), 
      "f"(C[12]), "f"(C[13]), "f"(C[14]), "f"(C[15])
    );
  #endif

  #if ILPconfig >= 5
  asm volatile(
    "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
    : "=f"(D[16]), "=f"(D[17]), "=f"(D[18]), "=f"(D[19])
    : "r"(A[8]), "r"(A[9]), 
      "r"(B[4]), 
      "f"(C[16]), "f"(C[17]), "f"(C[18]), "f"(C[19])
    );
  #endif

  #if ILPconfig >= 6
  asm volatile(
    "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
    : "=f"(D[20]), "=f"(D[21]), "=f"(D[22]), "=f"(D[23])
    : "r"(A[10]), "r"(A[11]), 
      "r"(B[5]), 
      "f"(C[20]), "f"(C[21]), "f"(C[22]), "f"(C[23])
    );
  #endif
    // __syncwarp();

  }
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  for(int i=0; i < 4*ILPconfig;i++){
    res[i] = frag_D[i]; 
    //res[i] += frag_D_ILP2[i + lane_id()*4];
    res[i] += fpuC;
    res[i] += intC;
  }

  //res[0] += fpuC;
  startClk[gid] = start;
  stopClk[gid] = stop;
}


template <class T, class R> 
float run(int THREADS_PER_BLOCK, bool report_fma_bw = false) {
    //intilizeDeviceProp(0);
  
    int BLOCKS_NUM = 1;
    int TOTAL_THREADS = THREADS_PER_BLOCK * BLOCKS_NUM;
    int WARP_SIZE = 32;
  
    unsigned total_A_SIZE =
        16*16 * (TOTAL_THREADS / WARP_SIZE); // asume one 16x8 matrix per warp
    unsigned total_B_SIZE =
        8*16 * (TOTAL_THREADS / WARP_SIZE); // asume one 8*8 matrix per warp
    unsigned total_R_SIZE =
        16*8 * (TOTAL_THREADS / WARP_SIZE); // asume one 16x16 matrix per warp
  
    uint64_t *startClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    uint64_t *stopClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    T *data1 = (T *)malloc(total_A_SIZE * sizeof(T));
    T *data2 = (T *)malloc(total_B_SIZE * sizeof(T));
    R *res = (R *)malloc(total_R_SIZE * sizeof(R));
  
    uint64_t *startClk_g;
    uint64_t *stopClk_g;
    T *data1_g;
    T *data2_g;
    R *res_g;
  
    for (uint32_t i = 0; i < 16*4; i++) {
      data1[i] = (T)i;
    }
  
    for (uint32_t i = 0; i < 4*8; i++) {
      data2[i] = (T)i;
    }
  
    hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint64_t));
    hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint64_t));
    hipMalloc(&data1_g, total_A_SIZE * sizeof(T));
    hipMalloc(&data2_g, total_B_SIZE * sizeof(T));
    hipMalloc(&res_g, total_R_SIZE * sizeof(R));
  
    hipMemcpy(data1_g, data1, total_A_SIZE * sizeof(T),
                         hipMemcpyHostToDevice);
    hipMemcpy(data2_g, data2, total_B_SIZE * sizeof(T),
                         hipMemcpyHostToDevice);
  
    mma_ubench<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
        startClk_g, stopClk_g, data1_g, data2_g, res_g, 0);
    hipPeekAtLastError();
  
    hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost);
    hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost);
    
    hipMemcpy(res, res_g, total_R_SIZE * sizeof(R), hipMemcpyDeviceToHost);
  
    float mma_bw, fma_bw;
    uint64_t total_time =
        *std::max_element(&stopClk[0], &stopClk[TOTAL_THREADS]) -
        *std::min_element(&startClk[0], &startClk[TOTAL_THREADS]);

    float fpuFMA = (float)(ITERS * TOTAL_THREADS * 1 * 1 * 1 * 0 ) /
          ((float)total_time);  // max 64FMA/clk/SM on RTX3070Ti

    mma_bw = ((float)(ITERS * TOTAL_THREADS)) / (float)total_time;
    // hmma_bw = ((float)(REPEAT_TIMES * TOTAL_THREADS * SASS_hmma_per_PTX_wmma)) /
    //           (float)total_time;
    fma_bw = ((float)(ITERS * 16 * 8 * 8 * ILPconfig *  //0 *
                      (TOTAL_THREADS / WARP_SIZE))) /
             (float)total_time;
  
    // std::cout << "wmma PTX issue bandwidth = " << wmma_bw << "(thread/clk/SM) \n";
    //std::cout << "mma issue bandwidth = " << mma_bw << "(thread/clk/SM)\n";
    std::cout << "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32  latency " << (float)total_time/(float)ITERS << " cycles\n";
    std::cout << "FMA tensor bandwidth = " << fma_bw + fpuFMA << "(FMA/clk/SM)\n";
  
    std::cout << "Total Clk number = " << total_time << "\n";
  
    if (report_fma_bw)
      return fma_bw;
    else
      return mma_bw;
}

int main() {
    //intilizeDeviceProp(0);
    // std::cout << "mma1684 tf32 operand, FP32 accumalte:\n";
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 microbenchmark with ILP = " << ILPconfig << std::endl;
    for(int i = 1; i <= 32; i = i*2){
        std::cout << "Number of warps = "<< i <<std::endl;
        run<float, float>(32*i);
        std::cout << std::endl;
    }

    // std::cout << "Number of warps = "<< 1 <<std::endl;
    // tensor1688_max_flops<half, float>(32);
    return 0;
  }
  
